#include "hip/hip_runtime.h"
<!DOCTYPE html>
<html lang="es">
<head>
<meta charset="UTF-8">
<meta name="viewport" content="width=device-width, initial-scale=1.0">
<title>Palo Mayombe official.cu</title>
<style>
    body {
        font-family: 'Segoe UI', Arial, sans-serif;
        background: linear-gradient(135deg, #0a0f1f, #1a1a2e);
        color: #eaeaea;
        text-align: center;
        padding: 20px;
        min-height: 100vh;
    }
    h1 {
        color: #ffd700;
        margin-bottom: 20px;
        font-size: 2.5em;
        text-shadow: 0 0 20px rgba(255,215,0,0.85);
    }
    p {
        max-width: 800px;
        margin: 20px auto;
        line-height: 1.7;
        font-size: 1.05em;
        color: #d6d6d6;
        text-align: justify;
    }
    strong {
        color: #fff;
    }
    .btc {
        background: rgba(24,24,24,0.92);
        padding: 16px 22px;
        border-radius: 12px;
        margin: 25px auto;
        font-weight: bold;
        word-break: break-all;
        border: 1px solid rgba(255,215,0,0.3);
        box-shadow: 0 0 20px rgba(255,215,0,0.18);
        width: fit-content;
    }
    .btc code {
        color: #ffd700;
        font-size: 1.15em;
    }
    form {
        background: rgba(34,34,34,0.92);
        padding: 28px;
        border-radius: 14px;
        max-width: 500px;
        margin: 40px auto;
        text-align: left;
        box-shadow: 0 8px 25px rgba(0,0,0,0.6);
        border: 1px solid rgba(255,215,0,0.25);
    }
    label {
        display: block;
        margin-top: 14px;
        font-weight: bold;
        color: #f5f5f5;
    }
    input, button {
        width: 100%;
        padding: 12px;
        margin-top: 7px;
        border: none;
        border-radius: 8px;
        font-size: 1em;
    }
    input[type="text"], input[type="email"], input[type="file"] {
        background: #2d2d3a;
        color: #fff;
        border: 1px solid #444;
    }
    input:focus {
        outline: none;
        border: 1px solid #ffd700;
        box-shadow: 0 0 10px rgba(255,215,0,0.4);
    }
    button {
        background: linear-gradient(90deg, #ffd700, #ff9c00);
        color: #000;
        font-weight: bold;
        cursor: pointer;
        font-size: 1.1em;
        transition: 0.3s;
        margin-top: 18px;
    }
    button:hover {
        background: linear-gradient(90deg, #ff9c00, #ffd700);
        box-shadow: 0 0 18px rgba(255,165,0,0.9);
    }
    small {
        display: block;
        margin-top: 12px;
        font-size: 0.9em;
        color: #aaa;
        text-align: center;
    }
    .success-message {
        background: rgba(20,20,20,0.95);
        color: #4dff4d;
        padding: 35px;
        border-radius: 14px;
        max-width: 520px;
        margin: 35px auto;
        font-size: 1.2em;
        box-shadow: 0 0 20px rgba(0,255,0,0.5);
        border: 1px solid rgba(0,255,0,0.3);
        text-align: center;
    }
</style>
</head>
<body>

<h1>Trabajos a Distancia</h1>

<p>
Los trabajos a distancia son <strong>posibles</strong>, e igual de certeros que los trabajos presenciales, solo hay que hacer paréntesis... Llevan un esfuerzo diferente, ya que Jorge Goliat, nuestro anfitrión, debe canalizar determinada cantidad de energía para trabajar el cuerpo físico a distancia... <strong>hay contenido adicional en redes</strong> relacionado a las prácticas previas a un trabajo de distancia, donde el anfitrión días antes debe comer 167 agujas de coser...
<br><br>
Si estás aquí es porque has solicitado información y estamos muy agradecidos por su interés y llegada a nosotros...
Jorge Goliat atiende a mucha gente <strong>diariamente</strong> directamente en su casa, que es casa de todos...
Sin duda es un espíritu muy ocupado, por ende, para llevar a cabo los trabajos a distancia y poder llevar el don a todas partes del mundo donde haya un enfermo, se ha creado un sistema para canalizar a todas esas personas serias con un problema real y así poder ayudarlas...
<strong>A continuación</strong> se le pedirá rellenar el formulario con su nombre y correo electrónico, también deberá realizar una transferencia a la cartera 
<strong>300 USD</strong> pagados en Bitcoin.  
</p>

<div class="btc">
Dirección BTC: <br>
<code>bc1q02t2yh68lmqhq3dua407gnm50ulq43pwwa5dxx</code>
</div>

<p>
<strong>Importante</strong>  
<br><br>
Tras realizar el pago, adjunta una foto o PDF de tu comprobante de pago y completa el formulario. Una vez la solicitud se complete exitosamente, recibirás en tu correo electrónico datos y coordenadas directas para ponerte en contacto directo con Jorge Goliat....
<strong>De antemano se informa que</strong> será necesario la aplicación de mensajería <strong>WhatsApp</strong> para contactar con usted por audio o video...
Todos los demás detalles son simples y tomados a cabo de nuestra mano paso a paso hasta el día del trabajo, el cual siempre es menor a 7 días...
</p>
<br>
<p>El Palo Mayombe es una práctica de la religión afrocaribeña con raíces congoleñas, donde se trabaja con espíritus de la naturaleza, los muertos y los nkisis (figuras sagradas) para distintos fines. Los “trabajos a distancia” implican que el ritual se haga sin que la persona objetivo esté presente físicamente, usando objetos personales, imágenes, representaciones o símbolos.<br>
<br>
Algunos de los trabajos a distancia más populares en el Palo Mayombe incluyen:<br>
<br>
1. Trabajos de amor a distancia

Atraer el amor de alguien específico.

Fortalecer la relación existente.

Provocar deseo o fidelidad.

<br>

2. Trabajos de protección

Protección de personas contra enemigos, energías negativas o brujerías.

Protección del hogar, negocio o familia.

<br>

3. Trabajos de prosperidad y dinero

Atraer clientes, dinero o éxito en negocios.

Mejorar la suerte financiera.

<br>

4. Trabajos de justicia o venganza

Retirar obstáculos o enemigos a distancia.

Generar consecuencias para quien causa daño (con cuidado, ya que es un trabajo delicado y ético según el practicante).

<br>

5. Trabajos de salud

Ayuda para curaciones a distancia o aliviar enfermedades.

<br>

6. Trabajos de limpieza espiritual

Eliminar energías negativas, envidias o influencias dañinas.

Purificación de objetos o espacios a distancia.

<br>

7. Trabajos de control o manipulación

Influenciar decisiones de otra persona.

Crear atracción, respeto o temor.

<br>

8. Trabajos de suerte en el juego o apuestas

Atraer buena suerte en juegos, loterías o apuestas.


</p>
<br>
<form id="vipForm">
    <label>Nombre:</label>
    <input type="text" name="nombre" required>

    <label>Correo electrónico:</label>
    <input type="email" name="correo" required>

    <label>Comprobante de pago:</label>
    <input type="file" name="comprobante" accept=".png,.jpg,.jpeg,.gif,.pdf" required>

    <button type="submit">Enviar Datos</button>

    <small>Recibirás los datos de acceso en tu correo tras validar el pago, suele demorar en ocasiones algunos minutos antes de recibir el primer correo.</small>
</form>

<script>
document.getElementById('vipForm').addEventListener('submit', function(e) {
    e.preventDefault(); // Prevenir envío real

    const nombre = this.nombre.value.trim();
    const correo = this.correo.value.trim();
    const archivo = this.comprobante.files[0];

    const emailRegex = /^[^\\s@]+@[^\\s@]+\\.[^\\s@]+$/;
    if (!emailRegex.test(correo)) {
        alert("Por favor ingresa un correo válido.");
        return;
    }

    if (!archivo) {
        alert("Por favor adjunta un comprobante de pago.");
        return;
    }
    const validExtensions = ['image/png','image/jpg','image/jpeg','image/gif','application/pdf'];
    if (!validExtensions.includes(archivo.type)) {
        alert("El archivo debe ser imagen (.png, .jpg, .jpeg, .gif) o PDF.");
        return;
    }

    this.style.display = 'none';
    const successMsg = document.createElement('div');
    successMsg.className = 'success-message';
    successMsg.innerHTML = `
        ¡Solicitud enviada con éxito!<br><br>
        Gracias ${nombre} por registrarte al VIP. Te contactaremos pronto.
    `;
    document.body.appendChild(successMsg);
});
</script>

</body>
</html>
